#include <iostream>
#include <thrust/device_vector.h>

#include <amgcl/amgcl.hpp>
#include <amgcl/make_solver.hpp>

#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/cusparse_ilu0.hpp>
#include <amgcl/solver/bicgstab.hpp>
#include <amgcl/backend/cuda.hpp>
#include <amgcl/adapter/crs_tuple.hpp>

#include <amgcl/profiler.hpp>

#include "sample_problem.hpp"

namespace amgcl {
    profiler<> prof("cuda");
}

int main(int argc, char *argv[]) {
    const int m = argc > 1 ? atoi(argv[1]) : 64;

    std::vector<int>    ptr, col;
    std::vector<double> val, rhs;

    using amgcl::prof;

    // 3d poisson in m*m*m cube:
    prof.tic("assemble");
    int n = sample_problem(m, val, col, ptr, rhs);
    prof.toc("assemble");

    // Setup solver:
    typedef amgcl::backend::cuda<double> Backend;
    typedef amgcl::make_solver<
        amgcl::amg<
            Backend,
            amgcl::coarsening::smoothed_aggregation,
            amgcl::relaxation::ilu0
            >,
        amgcl::solver::bicgstab< Backend >
        > Solver;

    // Init CUSPARSE (once per program lifespan):
    Solver::params  sprm;
    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    prof.tic("setup");
    Solver solve( boost::tie(n, ptr, col, val), sprm, bprm );
    prof.toc("setup");

    std::cout << solve.precond() << std::endl;

    // Solve the problem. The rhs and the solution vectors are in GPU memory.
    thrust::device_vector<double> f = rhs;
    thrust::device_vector<double> x(n);
    thrust::fill(x.begin(), x.end(), 0.0); // Initial approximation.

    int    iters;
    double error;

    prof.tic("solve");
    boost::tie(iters, error) = solve(f, x);
    prof.toc("solve");

    std::cout
        << "Iterations: " << iters << std::endl
        << "Error:      " << error << std::endl
        ;

    std::cout << prof << std::endl;
}

